#include "hip/hip_runtime.h"
#include "parser.h"
//#include "matcher.h"

// Considering the size of NFAs in application, each block handle a string's match individually

__global__ void matcher(u8 *states, u8 *final_states, int *begin_index_of_states, 
    int *pre_states, int *begin_index_of_pre, int state_num,
    int transition_num, u8 *str, int length, bool *matcher_result){
    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int thread_cnt = blockDim.x;
    int block_cnt = gridDim.x;

    int ind = 0, ch;
    int from, to;
    int pre_from, pre_to;

    //__shared__ u8 shared_state[2][(state_num-1)/(8*sizeof(u8)) + 1];
    //__shared__ u8 shared_final_states[(state_num-1)/(8*sizeof(u8)) + 1];

    __shared__ u8 shared_state[2][1<<13]; // state num should be less than 64K
    __shared__ u8 shared_final_states[1<<13]; // state num should be less than 64K 

    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
        shared_state[0][i] = 0;
        shared_state[1][i] = states[i];
        shared_final_states[i] = final_states[i];
    }
    __syncthreads();

    while(ind<length){
        ch = str[ind];
        from = begin_index_of_states[ch];
        if (ch<255) to = begin_index_of_states[ch+1];
        else to = state_num;

        for(int i = thread_idx+from; i<to; i+=thread_cnt){
            pre_from = begin_index_of_pre[i];
            if (i<state_num-1) pre_to = begin_index_of_pre[i+1];
            else pre_to = transition_num;
            int tmp = 0;
            for(int j=pre_from; j<pre_to; j++){
                tmp |= (shared_state[(ind+1)%2][pre_states[j]/(sizeof(u8)*8)] & (1<<(pre_states[j]%(sizeof(u8)*8))));
            }
            if(tmp) shared_state[ind%2][i/(sizeof(u8)*8)] |= (1<<(i%(sizeof(u8)*8))); // should be done with atomic operations
        }
        __syncthreads();
        for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
            shared_state[(ind+1)%2][i] = 0;
        }
        __syncthreads();
        ind+=1;
    }
    ind -= 1;
    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
        if (shared_state[ind%2][i]&  shared_final_states[i]) {*matcher_result = true;}
    }
    __syncthreads();
    return;
}


vector<int> gpu_matcher(int state_num, int transition_num, u8 *states, u8 *final_states, int *begin_index_of_states,
    int *begin_index_of_pre, int *pre_states, u8 *str, int length){
    vector<int> ret;
    ret.clear();

    u8 *d_states;
    u8 *d_final_states;
    int *d_begin_index_of_states;
    int *d_pre_states;
    int *d_begin_index_of_pre;
    bool *matcher_result = new bool;
    bool *d_matcher_result;
    *matcher_result = false;
    u8 *d_str;

    hipMalloc((void **)&d_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_final_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_begin_index_of_states, sizeof(int)*(256));
    hipMalloc((void **)&d_begin_index_of_pre, sizeof(int)*(state_num));
    hipMalloc((void **)&d_pre_states, sizeof(int)*(transition_num));
    hipMalloc((void **)&d_str, sizeof(u8)*length);
    hipMalloc((void **)&d_matcher_result, sizeof(bool));

    hipMemcpy(d_states, states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_final_states, final_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_states, begin_index_of_states, sizeof(int)*(256), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_pre, begin_index_of_pre, sizeof(int)*(state_num), hipMemcpyHostToDevice);
    hipMemcpy(d_pre_states, pre_states, sizeof(int)*(transition_num), hipMemcpyHostToDevice);
    hipMemcpy(d_str, str, sizeof(u8)*length, hipMemcpyHostToDevice);
    hipMemcpy(d_matcher_result, matcher_result, sizeof(bool), hipMemcpyHostToDevice);
    dim3 grid(1,0,0);
    dim3 block(1024,0,0);
    matcher<<<grid, block>>>(d_states, d_final_states, d_begin_index_of_states, d_pre_states, d_begin_index_of_pre,
            state_num, transition_num, d_str, length, d_matcher_result);
    hipDeviceSynchronize();

    hipMemcpy(matcher_result, d_matcher_result, sizeof(bool), hipMemcpyDeviceToHost);

    if(*matcher_result) ret.push_back(length-1);
    return ret;
}
