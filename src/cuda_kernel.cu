#include "hip/hip_runtime.h"
#include "parser.h"
//#include "matcher.h"

// Considering the size of NFAs in application, each block handle a string's match individually

__global__ void matcher(u8 *states, u8 *final_states, int *begin_index_of_states, 
    int *pre_states, int *begin_index_of_pre, int state_num,
    int transition_num, int str_num, u8 *str, int *str_begin_index, int *length, bool *matcher_result){
    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int thread_cnt = blockDim.x;
    int block_cnt = gridDim.x;

    int ind = 0, ch;
    int from, to;
    int pre_from, pre_to;

    //__shared__ u8 shared_state[2][(state_num-1)/(8*sizeof(u8)) + 1];
    //__shared__ u8 shared_final_states[(state_num-1)/(8*sizeof(u8)) + 1];

    __shared__ int shared_state[2][1<<10]; // state num should be less than 32K
    __shared__ int shared_final_states[1<<10]; // state num should be less than 32K

    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
        shared_final_states[i] = (int(final_states[4*i])) | (int(final_states[4*i+1])<<8) | (int(final_states[4*i+2])<<16) | (int(final_states[4*i+3])<<24);
    }
    __syncthreads();

    for (int str_id = block_idx; str_id < str_num; str_id += block_cnt){
        ind = str_begin_index[str_id];
        for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
            shared_state[(ind)%2][i] = 0;
            shared_state[(ind+1)%2][i] = (int(states[4*i])) | (int(states[4*i+1])<<8) | (int(states[4*i+2])<<16) | (int(states[4*i+3])<<24);
        }
        __syncthreads();
        while(ind<str_begin_index[str_id] + length[str_id]){
            ch = str[ind];
            from = begin_index_of_states[ch];
            if (ch<255) to = begin_index_of_states[ch+1];
            else to = state_num;

            for(int i = thread_idx+from; i<to; i+=thread_cnt){
                pre_from = begin_index_of_pre[i];
                if (i<state_num-1) pre_to = begin_index_of_pre[i+1];
                else pre_to = transition_num;
                int tmp = 0;
                for(int j=pre_from; j<pre_to; j++){
                    tmp |= (shared_state[(ind+1)%2][pre_states[j]/(sizeof(int)*8)] & (1<<(pre_states[j]%(sizeof(int)*8))));
                }
                if(tmp) atomicOr(&shared_state[ind%2][i/(sizeof(int)*8)], (1<<(i%(sizeof(int)*8)))); // should be done with atomic operations
            }
            __syncthreads();
            for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
                shared_state[(ind+1)%2][i] = 0;
            }
            __syncthreads();
            ind+=1;
        }
        ind -= 1;
        for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
            if (shared_state[ind%2][i] & shared_final_states[i]) {matcher_result[str_id] = true;}
        }
    }
    __syncthreads();
    return;
}


vector<int> gpu_matcher(int state_num, int transition_num, u8 *states, u8 *final_states, int *begin_index_of_states,
    int *begin_index_of_pre, int *pre_states, int str_num, u8 *str, int *str_begin_index, int *length){

    struct timeval start_time, end_time;
    vector<int> ret;
    ret.clear();

    states[0] = 1;

    u8 *d_states;
    u8 *d_final_states;
    int *d_begin_index_of_states;
    int *d_pre_states;
    int *d_begin_index_of_pre;
    bool *matcher_result = new bool [str_num];
    memset(matcher_result, 0, sizeof(bool)*str_num);
    bool *d_matcher_result;
    u8 *d_str;
    int *d_str_begin_index;
    int *d_length;

    int total_len = 0;
    for(int i=0; i<str_num; i++) total_len+=length[i];

    hipMalloc((void **)&d_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_final_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_begin_index_of_states, sizeof(int)*(256));
    hipMalloc((void **)&d_begin_index_of_pre, sizeof(int)*(state_num));
    hipMalloc((void **)&d_pre_states, sizeof(int)*(transition_num));
    hipMalloc((void **)&d_str, sizeof(u8)*total_len);
    hipMalloc((void **)&d_matcher_result, sizeof(bool)*str_num);
    hipMalloc((void **)&d_str_begin_index, sizeof(int)*str_num);
    hipMalloc((void **)&d_length, sizeof(int)*str_num);

    hipMemcpy(d_states, states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_final_states, final_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_states, begin_index_of_states, sizeof(int)*(256), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_pre, begin_index_of_pre, sizeof(int)*(state_num), hipMemcpyHostToDevice);
    hipMemcpy(d_pre_states, pre_states, sizeof(int)*(transition_num), hipMemcpyHostToDevice);
    
    gettimeofday(&start_time, NULL);
    hipMemcpy(d_str, str, sizeof(u8)*total_len, hipMemcpyHostToDevice);
    hipMemcpy(d_str_begin_index, str_begin_index, sizeof(int)*str_num, hipMemcpyHostToDevice);
    hipMemcpy(d_length, length, sizeof(int)*str_num, hipMemcpyHostToDevice);
    gettimeofday(&end_time, NULL);
    float elapsed_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 +
                         (end_time.tv_usec - start_time.tv_usec) / 1000.0;
    cout << "Cuda Mem Alloc Time Cost: " << elapsed_time << " ms\n";

    hipMemcpy(d_matcher_result, matcher_result, sizeof(bool), hipMemcpyHostToDevice);

    dim3 grid(64,1,1);
    dim3 block(1024,1,1);
    gettimeofday(&start_time, NULL);
    matcher<<<grid, block>>>(d_states, d_final_states, d_begin_index_of_states, d_pre_states, d_begin_index_of_pre,
            state_num, transition_num, str_num, d_str, d_str_begin_index, d_length, d_matcher_result);
    hipDeviceSynchronize();
    gettimeofday(&end_time, NULL);
    elapsed_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 +
                   (end_time.tv_usec - start_time.tv_usec) / 1000.0;
    cout << "Execute "<< total_len <<"Byte, Regex-Matcher Kernel Execution Time Cost: " << elapsed_time << " ms\n";

    hipMemcpy(matcher_result, d_matcher_result, sizeof(bool)*str_num, hipMemcpyDeviceToHost);
    
    hipFree(d_str);
    hipFree(d_states);
    hipFree(d_pre_states);
    hipFree(d_matcher_result);
    hipFree(d_final_states);
    hipFree(d_begin_index_of_states);
    hipFree(d_begin_index_of_pre);

    for (int i=0;i<str_num; i++) {
        if(matcher_result[i]) ret.push_back(1);
        else ret.push_back(0);
    }

    delete [] matcher_result;

    return ret;
}
