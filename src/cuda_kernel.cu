#include "hip/hip_runtime.h"
#include "matcher.h"

// Considering the size of NFAs in application, each block handle a string's match individually

__global__ void matcher(u8 *states, u8 *final_states, int *begin_index_of_states, 
    int *pre_states, int *begin_index_of_pre, int state_num,
    int transition_num, u8 *str, int length, bool *matcher_result){
    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int thread_cnt = blockDim.x;
    int block_cnt = gridDim.x;

    int ind = 0, ch;
    int from, to;
    int pre_from, pre_to;

    __shared__ u8 shared_state[2][(state_num-1)/(8*sizeof(u8)) + 1];
    __shared__ u8 shared_final_states[(state_num-1)/(8*sizeof(u8)) + 1];

    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
        shared_state[1][i] = states[i];
        shared_final_states[i] = final_states[i];
    }
    __syncthreads();

    while(ind<length){
        ch = str[ind];
        from = begin_index_of_states[ch];
        if (ch<255) to = begin_index_of_states[ch+1];
        else to = state_num;

        for(int i = thread_idx+from; i<to; i+=thread_cnt){
            pre_from = begin_index_of_pre[i];
            if (i<state_num-1) pre_to = begin_index_of_pre[i+1];
            else pre_to = transition_num;
            int tmp = 0;
            for(int j=pre_from; j<pre_to; j++){
                tmp |= (shared_state[(ind+1)%2][pre_states[j]/(sizeof(u8)*8)] & (1<<(pre_states[j]%(sizeof(u8)*8))));
            }
            if(tmp) shared_state[ind%2][pre_states[j]/(sizeof(u8)*8)] |= (1<<(pre_states[j]%(sizeof(u8)*8))); // should be done with atomic operations
        }
        __syncthreads();
        for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
            shared_state[(ind+1)%2][i] = 0;
        }
        __syncthreads();
        ind+=1;
    }
    ind -= 1;
    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
        if (shared_state[ind%2][i]&  shared_final_states[i]) {*matcher_result = true;}
    }
    __syncthreads();
    return;
}


vector<int> NFA_Matcher::gpu_matcher(u8 *str, int length){
    vector<int> ret;
    ret.clear();

    u8 *d_states;
    u8 *d_final_states;
    int *d_begin_index_of_states;
    int *d_pre_states;
    int *d_begin_index_of_pre;

    hipMalloc((void **)&d_states, , );

    return ret;
}
