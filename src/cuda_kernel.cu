#include "hip/hip_runtime.h"
#include "parser.h"
//#include "matcher.h"

// Considering the size of NFAs in application, each block handle a string's match individually

__global__ void matcher(u8 *states, u8 *final_states, int *begin_index_of_states, 
    int *pre_states, int *begin_index_of_pre, int state_num,
    int transition_num, u8 *str, int length, bool *matcher_result){
    int thread_idx = threadIdx.x;
    //int block_idx = blockIdx.x;
    int thread_cnt = blockDim.x;
    //int block_cnt = gridDim.x;

    int ind = 0, ch;
    int from, to;
    int pre_from, pre_to;

    //__shared__ u8 shared_state[2][(state_num-1)/(8*sizeof(u8)) + 1];
    //__shared__ u8 shared_final_states[(state_num-1)/(8*sizeof(u8)) + 1];

    __shared__ int shared_state[2][1<<10]; // state num should be less than 32K
    __shared__ int shared_final_states[1<<10]; // state num should be less than 32K

    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
        shared_state[0][i] = 0;
        shared_state[1][i] = (int(states[4*i])) | (int(states[4*i+1])<<8) | (int(states[4*i+2])<<16) | (int(states[4*i+3])<<24);
        shared_final_states[i] = (int(final_states[4*i])) | (int(final_states[4*i+1])<<8) | (int(final_states[4*i+2])<<16) | (int(final_states[4*i+3])<<24);
    }
    __syncthreads();

    while(ind<length){
        ch = str[ind];
        from = begin_index_of_states[ch];
        if (ch<255) to = begin_index_of_states[ch+1];
        else to = state_num;

        for(int i = thread_idx+from; i<to; i+=thread_cnt){
            pre_from = begin_index_of_pre[i];
            if (i<state_num-1) pre_to = begin_index_of_pre[i+1];
            else pre_to = transition_num;
            int tmp = 0;
            for(int j=pre_from; j<pre_to; j++){
                tmp |= (shared_state[(ind+1)%2][pre_states[j]/(sizeof(int)*8)] & (1<<(pre_states[j]%(sizeof(int)*8))));
            }
            if(tmp) atomicOr(&shared_state[ind%2][i/(sizeof(int)*8)], (1<<(i%(sizeof(int)*8)))); // should be done with atomic operations
        }
        __syncthreads();
        for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
            shared_state[(ind+1)%2][i] = 0;
        }
        __syncthreads();
        ind+=1;
    }
    ind -= 1;
    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(int)) + 1; i+=thread_cnt){
        if (shared_state[ind%2][i] & shared_final_states[i]) {*matcher_result = true;}
    }
    __syncthreads();
    return;
}


vector<int> gpu_matcher(int state_num, int transition_num, u8 *states, u8 *final_states, int *begin_index_of_states,
    int *begin_index_of_pre, int *pre_states, u8 *str, int length){

    struct timeval start_time, end_time;
    vector<int> ret;
    ret.clear();

    states[0] = 1;

    u8 *d_states;
    u8 *d_final_states;
    int *d_begin_index_of_states;
    int *d_pre_states;
    int *d_begin_index_of_pre;
    bool *matcher_result = new bool;
    bool *d_matcher_result;
    *matcher_result = false;
    u8 *d_str;

    hipMalloc((void **)&d_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_final_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_begin_index_of_states, sizeof(int)*(256));
    hipMalloc((void **)&d_begin_index_of_pre, sizeof(int)*(state_num));
    hipMalloc((void **)&d_pre_states, sizeof(int)*(transition_num));
    hipMalloc((void **)&d_str, sizeof(u8)*length);
    hipMalloc((void **)&d_matcher_result, sizeof(bool));

    hipMemcpy(d_states, states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_final_states, final_states, sizeof(u8)*((state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_states, begin_index_of_states, sizeof(int)*(256), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_pre, begin_index_of_pre, sizeof(int)*(state_num), hipMemcpyHostToDevice);
    hipMemcpy(d_pre_states, pre_states, sizeof(int)*(transition_num), hipMemcpyHostToDevice);
    
    gettimeofday(&start_time, NULL);
    hipMemcpy(d_str, str, sizeof(u8)*length, hipMemcpyHostToDevice);
    gettimeofday(&end_time, NULL);
    float elapsed_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 +
                         (end_time.tv_usec - start_time.tv_usec) / 1000.0;
    cout << "Cuda Mem Alloc Time Cost: " << elapsed_time << " ms\n";

    hipMemcpy(d_matcher_result, matcher_result, sizeof(bool), hipMemcpyHostToDevice);

    dim3 grid(1,1,1);
    dim3 block(1024,1,1);
    gettimeofday(&start_time, NULL);
    matcher<<<grid, block>>>(d_states, d_final_states, d_begin_index_of_states, d_pre_states, d_begin_index_of_pre,
            state_num, transition_num, d_str, length, d_matcher_result);
    hipDeviceSynchronize();
    gettimeofday(&end_time, NULL);
    elapsed_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 +
                   (end_time.tv_usec - start_time.tv_usec) / 1000.0;
    cout << "Execute "<< length <<"Byte, Regex-Matcher Kernel Execution Time Cost: " << elapsed_time << " ms\n";

    hipMemcpy(matcher_result, d_matcher_result, sizeof(bool), hipMemcpyDeviceToHost);
    
    hipFree(d_str);
    hipFree(d_states);
    hipFree(d_pre_states);
    hipFree(d_matcher_result);
    hipFree(d_final_states);
    hipFree(d_begin_index_of_states);
    hipFree(d_begin_index_of_pre);

    if(*matcher_result) ret.push_back(length-1);

    delete matcher_result;
    
    return ret;
}
