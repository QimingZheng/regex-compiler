#include "hip/hip_runtime.h"
#include "matcher.h"

// Considering the size of NFAs in application, each block handle a string's match individually

__global__ void matcher(u8 *states, u8 *final_states, int *begin_index_of_states, 
    int *pre_states, int *begin_index_of_pre, int state_num,
    int transition_num, u8 *str, int length, bool *matcher_result){
    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int thread_cnt = blockDim.x;
    int block_cnt = gridDim.x;

    int ind = 0, ch;
    int from, to;
    int pre_from, pre_to;

    __shared__ u8 shared_state[2][(state_num-1)/(8*sizeof(u8)) + 1];
    __shared__ u8 shared_final_states[(state_num-1)/(8*sizeof(u8)) + 1];

    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
        shared_state[1][i] = states[i];
        shared_final_states[i] = final_states[i];
    }
    __syncthreads();

    while(ind<length){
        ch = str[ind];
        from = begin_index_of_states[ch];
        if (ch<255) to = begin_index_of_states[ch+1];
        else to = state_num;

        for(int i = thread_idx+from; i<to; i+=thread_cnt){
            pre_from = begin_index_of_pre[i];
            if (i<state_num-1) pre_to = begin_index_of_pre[i+1];
            else pre_to = transition_num;
            int tmp = 0;
            for(int j=pre_from; j<pre_to; j++){
                tmp |= (shared_state[(ind+1)%2][pre_states[j]/(sizeof(u8)*8)] & (1<<(pre_states[j]%(sizeof(u8)*8))));
            }
            if(tmp) shared_state[ind%2][pre_states[j]/(sizeof(u8)*8)] |= (1<<(pre_states[j]%(sizeof(u8)*8))); // should be done with atomic operations
        }
        __syncthreads();
        for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
            shared_state[(ind+1)%2][i] = 0;
        }
        __syncthreads();
        ind+=1;
    }
    ind -= 1;
    for(int i = thread_idx; i < (state_num-1)/(8*sizeof(u8)) + 1; i+=thread_cnt){
        if (shared_state[ind%2][i]&  shared_final_states[i]) {*matcher_result = true;}
    }
    __syncthreads();
    return;
}


vector<int> gpu_matcher(NFA_Matcher &nfa_mat, u8 *str, int length){
    vector<int> ret;
    ret.clear();

    u8 *d_states;
    u8 *d_final_states;
    int *d_begin_index_of_states;
    int *d_pre_states;
    int *d_begin_index_of_pre;
    bool *matcher_result = new bool;
    bool *d_matcher_result;
    *matcher_result = false;
    u8 *d_str;

    hipMalloc((void **)&d_states, sizeof(u8)*((nfa_mat.state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_final_states, sizeof(u8)*((nfa_mat.state_num-1)/(8*sizeof(u8)) + 1));
    hipMalloc((void **)&d_begin_index_of_states, sizeof(int)*(256));
    hipMalloc((void **)&d_begin_index_of_pre, sizeof(int)*(nfa_mat.state_num));
    hipMalloc((void **)&d_pre_states, sizeof(int)*(nfa_mat.transition_num));
    hipMalloc((void **)&d_str, sizeof(u8)*length);
    hipMalloc((void **)&d_matcher_result, sizeof(bool));

    hipMemcpy(d_states, nfa_mat.states, sizeof(u8)*((nfa_mat.state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_final_states, nfa_mat.final_states, sizeof(u8)*((nfa_mat.state_num-1)/(8*sizeof(u8)) + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_states, nfa_mat.begin_index_of_states, sizeof(int)*(256), hipMemcpyHostToDevice);
    hipMemcpy(d_begin_index_of_pre, nfa_mat.begin_index_of_pre, sizeof(int)*(nfa_mat.state_num), hipMemcpyHostToDevice);
    hipMemcpy(d_pre_states, nfa_mat.pre_states, sizeof(int)*(nfa_mat.transition_num), hipMemcpyHostToDevice);
    hipMemcpy(d_str, str, sizeof(u8)*length, hipMemcpyHostToDevice);
    hipMemcpy(d_matcher_result, matcher_result, sizeof(bool), hipMemcpyHostToDevice);
    dim3 grid(1,0,0);
    dim3 block(1024,0,0);
    matcher<<<grid, block>>>(d_states, d_final_states, d_begin_index_of_states, d_pre_states, d_begin_index_of_pre, state_num, transition_num, d_str, length, matcher_result);
    hipDeviceSynchronize();

    if(*matcher_result) ret.push_back(length-1);
    return ret;
}
