#include "matcher.h"
#include "cuda_kernel.cu"

int main(){
    char regex[]="(asd|fns*daj)*|fsj(dav)*f(fs|da)";
    int length = strlen(regex);
    
    AST ast(regex, length);
    //ast.traverse();
    
    GlushKov_NFA NFA(&ast);
    //NFA.traverse();
    
    NFA_Matcher Matcher(&NFA);

    u8 input_str_0[] = "asfnsssdajsjdavdavffda";
    length = 22;

    vector<int> tmp = Matcher.naive_matcher(input_str_0, length);
    cout<<tmp.size()<<endl;
    tmp = Matcher.optimized_matcher(input_str_0, length);
    cout<<tmp.size()<<endl;
    tmp = gpu_matcher(Matcher.state_num, Matcher.transition_num, Matcher.states, Matcher.final_states, Matcher.begin_index_of_states,
        Matcher.begin_index_of_pre, Matcher.pre_states, input_str_0, length);
    cout<<tmp.size()<<endl;

    u8 input_str_1[] = "fsjdavdavdavdavdavdavdavffffda";
    length = 30;
    
    tmp = Matcher.naive_matcher(input_str_1, length);
    cout<<tmp.size()<<endl;
    tmp = Matcher.optimized_matcher(input_str_1, length);
    cout<<tmp.size()<<endl;
    tmp = gpu_matcher(Matcher.state_num, Matcher.transition_num, Matcher.states, Matcher.final_states, Matcher.begin_index_of_states,
        Matcher.begin_index_of_pre, Matcher.pre_states, input_str_1, length);
    cout<<tmp.size()<<endl;

    return 0;
}