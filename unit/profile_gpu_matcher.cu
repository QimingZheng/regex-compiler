#include "matcher.h"
#include "cuda_kernel.cu"

int main(int argc, char **argv){
    assert(argc == 3);
    char regex[]="(qq)*|(we)rtyu(io)*p*as*df((gh)|(jkl)|(zxc)|(vbn)|(m123)|(4567)|(890q)|(wert)y*ui)op*(a*s*)dfghjklzxcvbnm";
    int length = strlen(regex);
    
    AST ast(regex, length);
    GlushKov_NFA NFA(&ast);
    
    NFA_Matcher Matcher(&NFA);

    int str_num = atoi(argv[2]);
    u8 *input_str = new u8[atoi(argv[1])];
    int *str_len = new int[str_num];
    int *str_begin_ind = new int[str_num];

    for(int i=0;i<str_num-1;i++) {
        str_len[i] = atoi(argv[1])/str_num;
        str_begin_ind = i*(atoi(argv[1])/str_num);
    }
    str_len[str_num-1] = (atoi(argv[1])%str_num == 0)? atoi(argv[1])/str_num : atoi(argv[1])%str_num;
    str_begin_ind[str_num-1] = (str_num-1)*(atoi(argv[1])/str_num);

    gpu_matcher(Matcher.state_num, Matcher.transition_num, Matcher.states, Matcher.final_states, Matcher.begin_index_of_states,
        Matcher.begin_index_of_pre, Matcher.pre_states, str_num, input_str, str_begin_ind, str_len);

    return 0;
}